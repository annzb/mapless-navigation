#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <thread>
#include <math.h>
#include <Eigen/Core>
#include <mutex>

#include <hipfft/hipfft.h>
#include "coloradar_cuda.h"


double blackman(int i, int n) {
    double a0 = 0.42;
    double a1 = 0.5;
    double a2 = 0.08;
    return (a0 - a1 * cos((2.0 * M_PI * double(i)) / double(n)) + a2 * cos((4.0 * M_PI * double(i)) / double(n)));
}

template<typename T>
void checkCudaArray(T* device_array, size_t num_elements, std::string description) {
    std::cout << "Total elements in " << description << ": " << num_elements << std::endl;
    std::vector<T> host_data(num_elements);
    hipError_t err = hipMemcpy(host_data.data(), device_array, sizeof(T) * num_elements, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA Memcpy error (" << description << "): " << hipGetErrorString(err) << std::endl;
        return;
    }
    size_t non_zero_count = 0;
    for (size_t i = 0; i < num_elements; ++i) {
        if (host_data[i] != 0.0) {
            non_zero_count++;
        }
    }
    std::cout << "Non-zero elements in " << description << ": " << non_zero_count << std::endl << std::endl;
}


template<>
void checkCudaArray(hipDoubleComplex* device_array, size_t num_elements, std::string description) {
    std::cout << "Total elements in " << description << ": " << num_elements << std::endl;
    std::vector<hipDoubleComplex> host_data(num_elements);
    hipError_t err = hipMemcpy(host_data.data(), device_array, sizeof(hipDoubleComplex) * num_elements, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA Memcpy error (" << description << "): " << hipGetErrorString(err) << std::endl;
        return;
    }
    size_t non_zero_count = 0;
    for (size_t i = 0; i < num_elements; ++i) {
        if (host_data[i].x != 0.0 || host_data[i].y != 0.0) {
            non_zero_count++;
        }
    }
    std::cout << "Non-zero elements in " << description << ": " << non_zero_count << std::endl << std::endl;
}

template<typename T>
void cudaCopy(T* dest, std::vector<T> source) {
    hipError_t err = hipMemcpy(dest, source.data(), sizeof(T) * source.size(), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error:  " << hipGetErrorString(err) << std::endl;
    }
}


std::vector<float> coloradar::cubeToHeatmap(std::vector<int16_t> datacube, coloradar::RadarConfig* config) {
    bool collapse_doppler_ = true;  // WARNING: default false
    bool remove_antenna_coupling_ = true;  // WARNING: default true
    bool phase_freq_calib_ = true;  // WARNING: default false
    
    hipfftHandle range_plan_; // fft plan for range fft
    hipfftHandle doppler_plan_; // fft plan for doppler fft
    hipfftHandle angle_plan_; // fft plan for angle of arrival fft
    hipDoubleComplex* coupling_signature_; // coupling signature
    hipDoubleComplex* phase_calib_mat_; // phase calibration matrix
    hipDoubleComplex* freq_calib_mat_; // frequency calibration matrix
    hipDoubleComplex* range_fft_data_; // range fft data buffer
    hipDoubleComplex* doppler_fft_data_; // doppler fft data buffer
    hipDoubleComplex* angle_fft_data_; // angle fft data buffer
    double* range_window_func_; // range window function values
    double* doppler_window_func_; // doppler window function values
    double* az_window_func_; // azimuth window function values
    double* el_window_func_; // elevation window function values
    float* magnitudes_out_; // complex magnitudes of aoa fft output for publishing
    float* static_bins_; // only the static doppler bins of the steered output
    int16_t* int_frame_data_; // container for int-valued adc data from ros message
    int* virtualArrayMap;

    // Allocate datacube
    hipMalloc(&int_frame_data_, sizeof(int16_t) * datacube.size());
    cudaCopy(int_frame_data_, datacube);

    // Allocate virtual array map
    hipMalloc(&virtualArrayMap, sizeof(int) * 4 * config->numVirtualElements);
    cudaCopy(virtualArrayMap, config->virtualArrayMap);

    // Allocate memory for coupling signature
    hipMalloc(&coupling_signature_, sizeof(hipDoubleComplex) * config->numPosRangeBins * config->numTxAntennas * config->numRxAntennas);
    std::vector<hipDoubleComplex> h_couplingCalibMatrix(config->couplingCalibMatrix.size());
    for (size_t i = 0; i < config->couplingCalibMatrix.size(); ++i) {
        h_couplingCalibMatrix[i] = make_hipDoubleComplex(config->couplingCalibMatrix[i].real(), config->couplingCalibMatrix[i].imag());
    }
    cudaCopy(coupling_signature_, h_couplingCalibMatrix);
    // hipMemcpy(coupling_signature_, h_couplingCalibMatrix.data(), sizeof(hipDoubleComplex) * h_couplingCalibMatrix.size(), hipMemcpyHostToDevice);

    // Allocate memory for frequency calibration matrix
    hipMalloc(&freq_calib_mat_, sizeof(hipDoubleComplex) * config->numRangeBins * config->numTxAntennas * config->numRxAntennas);
    std::vector<hipDoubleComplex> h_freqCalibMatrix(config->calFrequencyCalibMatrix.size());
    for (size_t i = 0; i < config->calFrequencyCalibMatrix.size(); ++i) {
        h_freqCalibMatrix[i] = make_hipDoubleComplex(config->calFrequencyCalibMatrix[i].real(), config->calFrequencyCalibMatrix[i].imag());
    }
    hipMemcpy(freq_calib_mat_, h_freqCalibMatrix.data(), sizeof(hipDoubleComplex) * h_freqCalibMatrix.size(), hipMemcpyHostToDevice);

    // Allocate memory for phase calibration matrix
    hipMalloc(&phase_calib_mat_, sizeof(hipDoubleComplex) * config->numTxAntennas * config->numRxAntennas);
    std::vector<hipDoubleComplex> h_phaseCalibMatrix(config->calPhaseCalibMatrix.size());
    for (size_t i = 0; i < config->calPhaseCalibMatrix.size(); ++i) {
        h_phaseCalibMatrix[i] = make_hipDoubleComplex(config->calPhaseCalibMatrix[i].real(), config->calPhaseCalibMatrix[i].imag());
    }
    hipMemcpy(phase_calib_mat_, h_phaseCalibMatrix.data(), sizeof(hipDoubleComplex) * h_phaseCalibMatrix.size(), hipMemcpyHostToDevice);

    checkCudaArray(coupling_signature_, config->numPosRangeBins * config->numTxAntennas * config->numRxAntennas, "coupling_signature_");
    checkCudaArray(freq_calib_mat_, config->numRangeBins * config->numTxAntennas * config->numRxAntennas, "freq_calib_mat_");
    checkCudaArray(phase_calib_mat_, config->numTxAntennas * config->numRxAntennas, "phase_calib_mat_");

    int rank = 1;
    int angle_rank = 2;
    int n_range [1] = {config->numRangeBins};
    int n_doppler [1] = {config->numDopplerBins};
    int n_angle [2] = {config->numAzimuthBeams, config->numElevationBeams};
    int howmany_range = config->numTxAntennas * config->numRxAntennas * config->numDopplerBins;
    int howmany_doppler = config->numTxAntennas * config->numRxAntennas * config->numPosRangeBins;
    int howmany_angle = config->numPosRangeBins * config->numDopplerBins;
    int range_dist = config->numRangeBins;
    int doppler_dist = 1;
    int angle_dist = config->numAzimuthBeams * config->numElevationBeams;
    int range_stride = 1;
    int doppler_stride = config->numPosRangeBins * config->numTxAntennas * config->numRxAntennas;
    int angle_stride = 1;
    int *range_embed = n_range;
    int *doppler_embed = n_doppler;
    int *angle_embed = n_angle;
    hipMalloc(&range_fft_data_, sizeof(hipDoubleComplex) * config->numRangeBins * config->numDopplerBins * config->numTxAntennas * config->numRxAntennas);
    hipMalloc(&doppler_fft_data_, sizeof(hipDoubleComplex) * config->numPosRangeBins * config->numDopplerBins * config->numTxAntennas * config->numRxAntennas);
    hipMalloc(&angle_fft_data_, sizeof(hipDoubleComplex) * config->numPosRangeBins * config->numDopplerBins * config->numAzimuthBeams * config->numElevationBeams);
    hipMalloc(&magnitudes_out_, sizeof(float) * config->numPosRangeBins * config->numDopplerBins * config->numAzimuthBeams * config->numElevationBeams);
    if (collapse_doppler_)
        hipMalloc(&static_bins_, sizeof(float) * 2 * config->numPosRangeBins * config->numAzimuthBeams * config->numElevationBeams);
    hipfftPlanMany(&range_plan_, rank, n_range, range_embed, range_stride, range_dist, range_embed, range_stride, range_dist, HIPFFT_Z2Z , howmany_range);
    hipfftPlanMany(&doppler_plan_, rank, n_doppler, doppler_embed, doppler_stride, doppler_dist, doppler_embed, doppler_stride, doppler_dist, HIPFFT_Z2Z, howmany_doppler);
    hipfftPlanMany(&angle_plan_, angle_rank, n_angle, angle_embed, angle_stride, angle_dist, angle_embed, angle_stride, angle_dist, HIPFFT_Z2Z, howmany_angle);
    hipMalloc(&range_window_func_, sizeof(double) * config->numRangeBins);
    hipMalloc(&doppler_window_func_, sizeof(double) * config->numDopplerBins);
    hipMalloc(&az_window_func_, sizeof(double) * config->azimuthApertureLen);
    hipMalloc(&el_window_func_, sizeof(double) * config->elevationApertureLen);

    std::vector<double> range_window_local(config->numRangeBins);
    std::vector<double> doppler_window_local(config->numDopplerBins);
    std::vector<double> az_window_local(config->azimuthApertureLen);
    std::vector<double> el_window_local(config->elevationApertureLen);
    for (int range_idx = 0; range_idx < config->numRangeBins; range_idx++)
      range_window_local[range_idx] = blackman(range_idx, config->numRangeBins);
    for (int doppler_idx = 0; doppler_idx < config->numDopplerBins; doppler_idx++)
      doppler_window_local[doppler_idx] = blackman(doppler_idx, config->numDopplerBins);
    for (int az_idx = 0; az_idx < config->azimuthApertureLen; az_idx++)
      az_window_local[az_idx] = blackman(az_idx, config->azimuthApertureLen);
    for (int el_idx = 0; el_idx < config->elevationApertureLen; el_idx++)
      el_window_local[el_idx] = blackman(el_idx, config->elevationApertureLen);
    cudaCopy(range_window_func_, range_window_local);
    checkCudaArray(range_window_func_, config->numRangeBins, "range_window_func_");
    cudaCopy(doppler_window_func_, doppler_window_local);
    checkCudaArray(doppler_window_func_, config->numDopplerBins, "doppler_window_func_");
    cudaCopy(az_window_func_, az_window_local);
    checkCudaArray(az_window_func_, config->azimuthApertureLen, "az_window_func_");
    cudaCopy(el_window_func_, el_window_local);
    checkCudaArray(el_window_func_, config->elevationApertureLen, "el_window_func_");

    setFrameData(config->numRangeBins, config->numDopplerBins, config->numTxAntennas, config->numRxAntennas, int_frame_data_, range_fft_data_);
    if (phase_freq_calib_)
        applyPhaseFreqCal(config->numRangeBins, config->numDopplerBins, config->numTxAntennas, config->numRxAntennas, range_fft_data_, freq_calib_mat_, phase_calib_mat_);

    applyWindow(config->numRangeBins, 1, config->numRangeBins, config->numTxAntennas * config->numRxAntennas * config->numDopplerBins, range_window_func_, range_fft_data_);
    // run range fft
    hipfftExecZ2Z(range_plan_, range_fft_data_, range_fft_data_, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    // remove DC and negative frequency values from the range fft output
    removeNegSpectrum(config->numRangeBins, config->numPosRangeBins, config->numDopplerBins, config->numTxAntennas * config->numRxAntennas, range_fft_data_, doppler_fft_data_);
    checkCudaArray(range_fft_data_, config->numRangeBins * config->numDopplerBins * config->numTxAntennas * config->numRxAntennas, "range_fft_data_");
    
    if (remove_antenna_coupling_)
        removeCoupling(config->numPosRangeBins, config->numDopplerBins, config->numTxAntennas * config->numRxAntennas, doppler_fft_data_, coupling_signature_);
    // apply doppler window function
    applyWindow(1, config->numPosRangeBins * config->numTxAntennas * config->numRxAntennas, config->numDopplerBins, config->numTxAntennas * config->numRxAntennas * config->numPosRangeBins, doppler_window_func_, doppler_fft_data_);
    // run doppler fft
    hipfftExecZ2Z(doppler_plan_, doppler_fft_data_, doppler_fft_data_, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    checkCudaArray(doppler_fft_data_, config->numPosRangeBins * config->numDopplerBins * config->numTxAntennas * config->numRxAntennas, "doppler_fft_data_");

    // memset angle fft data back to zero
    // entries that are unset after array is filled with
    // samples will become zero padding
    hipMemset(angle_fft_data_, 0, sizeof(hipDoubleComplex) * config->numAzimuthBeams * config->numElevationBeams * config->numPosRangeBins * config->numDopplerBins);
    // move doppler fft result into angle fft data array
    // and apply azimuth and elevation window functions
    // not using the applyWindow kernel because it's not compatible
    // with the data layout required for the angle fft
    rearrangeData(config->numPosRangeBins, config->numDopplerBins, config->numTxAntennas, config->numRxAntennas, config->numAzimuthBeams, config->numElevationBeams, config->numVirtualElements, virtualArrayMap, az_window_func_, el_window_func_, doppler_fft_data_, angle_fft_data_);
    // run angle fft
    hipfftExecZ2Z(angle_plan_, angle_fft_data_, angle_fft_data_, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    checkCudaArray(angle_fft_data_, config->numPosRangeBins * config->numDopplerBins * config->numAzimuthBeams * config->numElevationBeams, "angle_fft_data_");

    // reorder data for publication
    // includes rearranging the doppler, azimuth, and elevation dimensions
    // so zero frequency is centered (fftshift in Matlab and SciPy)
    assembleMsg(config->numPosRangeBins, config->numDopplerBins, config->numAzimuthBeams, config->numElevationBeams, angle_fft_data_, magnitudes_out_);
    if (collapse_doppler_)
        collapseDoppler(config->numPosRangeBins, config->numDopplerBins, config->numAngles, config->dopplerBinWidth, magnitudes_out_, static_bins_);
    checkCudaArray(magnitudes_out_, config->numPosRangeBins * config->numDopplerBins * config->numAzimuthBeams * config->numElevationBeams, "magnitudes_out_");

    std::memcpy(config->elevationBins.data(), config->elevationAngles.data(), sizeof(float) * config->numElevationBeams);
    std::memcpy(config->azimuthBins.data(), config->azimuthAngles.data(), sizeof(float) * config->numAzimuthBeams);

    std::vector<float> image;
    if (collapse_doppler_) {
        image.resize(2 * config->numAngles * config->numPosRangeBins);
        hipMemcpy(&image[0], static_bins_, sizeof(float) * 2 * config->numPosRangeBins * config->numAngles, hipMemcpyDefault);
    } else {
        image.resize(config->numAngles * config->numPosRangeBins * config->numDopplerBins);
        hipMemcpy(&image[0], magnitudes_out_, sizeof(float) * config->numPosRangeBins * config->numDopplerBins * config->numAngles, hipMemcpyDefault);
    }

    hipFree(virtualArrayMap);
    hipFree(coupling_signature_);
    hipFree(phase_calib_mat_);
    hipFree(freq_calib_mat_);
    hipFree(range_window_func_);
    hipFree(doppler_window_func_);
    hipFree(az_window_func_);
    hipFree(el_window_func_);
    hipFree(range_fft_data_);
    hipFree(doppler_fft_data_);
    hipFree(angle_fft_data_);
    hipFree(magnitudes_out_);
    hipFree(int_frame_data_);
    if (collapse_doppler_) hipFree(static_bins_);

    return image;
}
